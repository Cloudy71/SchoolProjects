#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "pic_type.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale( CUDA_Pic colorPic, CUDA_Pic colorPic2 )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= colorPic.Size.y ) return;
	if ( x >= colorPic.Size.x ) return;

	// Get point from color picture
	uchar3 bgr = ( x < y*(colorPic.Size.y/ colorPic.Size.x)) ? colorPic.P_uchar3[ y * colorPic.Size.x + x ] : colorPic2.P_uchar3[y * colorPic.Size.x + x];
	printf("B=%d, G=%d, R=%d\n", bgr.x, bgr.y, bgr.z);

	// Store BW point to new image
	colorPic.P_uchar3[y * colorPic.Size.x + x] = bgr;
}

void cu_run_grayscale( CUDA_Pic colorPic, CUDA_Pic colorPic2)
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 16;
	dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_grayscale<<< blocks, threads >>>( colorPic, colorPic2);

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
	hipDeviceSynchronize();
}
