#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Manipulation with prepared image.
//
// ***********************************************************************

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "pic_type.h"

// Every threads identifies its position in grid and in block and modify image
__global__ void kernel_animation(CUDA_Pic cuda_pic)
{
	// X,Y coordinates 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= cuda_pic.Size.x) return;
	if (y >= cuda_pic.Size.y) return;
	uint2 xy[5] = { {125,130}, {250,130}, {375,130}, {190,190}, {315,190} };
	int outerRadius = 60;
	int innerRadius = 52;
	uchar3 bgr;

	//tanzania
	/*if (x<=(cuda_pic.Size.y-y)*1.5)
		bgr = { 58,181,30 };
	else
		bgr = {221,163,0};

	if ((x <= ((cuda_pic.Size.y - y)*1.5)+76) && (x > ((cuda_pic.Size.y - y)*1.5) - 76))
		bgr = { 22,209,252 };

	if ((x <= ((cuda_pic.Size.y - y)*1.5) + 52) && (x > ((cuda_pic.Size.y - y)*1.5) - 52))
		bgr = { 0,0,0 };*/

		//kuwait
		/*if (y < cuda_pic.Size.y / 3)
			bgr = { 61,122,0 };
		else if (y <= 2 * cuda_pic.Size.y / 3 && y >= cuda_pic.Size.y / 3)
			bgr = { 255,255,255 };
		else
			bgr = { 38,17,206 };

		if ((x <= y*1.5 && y < cuda_pic.Size.y / 3) || (x <= (cuda_pic.Size.y - y) * 1.5 && y > 2*cuda_pic.Size.y / 3)
			|| (x<=cuda_pic.Size.x/4 && y>= cuda_pic.Size.y / 3 && y <= 2 * cuda_pic.Size.y / 3))
			bgr = { 0,0,0 };*/

			//Great Britain
			/*bgr = { 125,36,0 };
			if ((x > cuda_pic.Size.x / 2 - 50 && x < cuda_pic.Size.x / 2 + 50) || (y > cuda_pic.Size.y / 2 - 50 && y < cuda_pic.Size.y / 2 + 50)
				 || (x <= ((cuda_pic.Size.y - y)*2.) + 60) && (x > ((cuda_pic.Size.y - y)*2.) - 60)
				 || (x <= ((y*2.) + 60) && (x > (y*2.) - 60)))
				bgr = { 255,255,255 };
			if ((x > cuda_pic.Size.x / 2 - 30 && x < cuda_pic.Size.x / 2 + 30) || (y > cuda_pic.Size.y / 2 - 30 && y < cuda_pic.Size.y / 2 + 30)
				|| (x <= ((cuda_pic.Size.y - y)*2.)) && (x >((cuda_pic.Size.y - y)*2.) - 40)
				|| (x <= ((y*2.)) && (x > (y*2.) - 40)))
				bgr = {43,20,207};*/

	bgr = { 255,255,255 };

	if ((outerRadius*outerRadius > (xy[0].x - x)*(xy[0].x - x) + (xy[0].y - y)*(xy[0].y - y)) && (innerRadius*innerRadius < (xy[0].x - x)*(xy[0].x - x) + (xy[0].y - y)*(xy[0].y - y)))
		bgr = { 199,133,0 };
	else if ((outerRadius*outerRadius >(xy[1].x - x)*(xy[1].x - x) + (xy[1].y - y)*(xy[1].y - y)) && (innerRadius*innerRadius < (xy[1].x - x)*(xy[1].x - x) + (xy[1].y - y)*(xy[1].y - y)))
		bgr = { 0,0,0 };
	else if ((outerRadius*outerRadius >(xy[2].x - x)*(xy[2].x - x) + (xy[2].y - y)*(xy[2].y - y)) && (innerRadius*innerRadius < (xy[2].x - x)*(xy[2].x - x) + (xy[2].y - y)*(xy[2].y - y)))
		bgr = { 36,0,223 };
	else if ((outerRadius*outerRadius >(xy[3].x - x)*(xy[3].x - x) + (xy[3].y - y)*(xy[3].y - y)) && (innerRadius*innerRadius < (xy[3].x - x)*(xy[3].x - x) + (xy[3].y - y)*(xy[3].y - y)))
		bgr = { 0,195,244 };
	else if ((outerRadius*outerRadius >(xy[4].x - x)*(xy[4].x - x) + (xy[4].y - y)*(xy[4].y - y)) && (innerRadius*innerRadius < (xy[4].x - x)*(xy[4].x - x) + (xy[4].y - y)*(xy[4].y - y)))
		bgr = { 61,159,0 };
	

	cuda_pic.P_uchar3[ y * cuda_pic.Size.x + x ] = bgr;
}

void cu_draw_flag( CUDA_Pic pic, uint2 block_size )
{
	hipError_t cerr;

	// Grid creation with computed organization
	dim3 grid( ( pic.Size.x + block_size.x - 1 ) / block_size.x, ( pic.Size.y + block_size.y - 1 ) / block_size.y );
	kernel_animation<<< grid, dim3( block_size.x, block_size.y ) >>>( pic );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();

}
